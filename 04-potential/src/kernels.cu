#include "hip/hip_runtime.h"
#include "kernels.h"

#define MAX(x,y) (x > y ? x : y)
#define EDGE_BLOCK 64
#define POINT_BLOCK 64


__global__ void simulate_compulsion(const Point<double>* points, const Edge<std::uint32_t>* edges, const std::uint32_t* lengths, Point<double>* velocities, double compulsion_constant, double vertex_mass, double time_quantum)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	Edge<std::uint32_t> edge = edges[idx];
	double x_diff =  points[edge.p2].x-points[edge.p1].x;
	double y_diff = points[edge.p2].y - points[edge.p1].y;
	double dist_squared = x_diff*x_diff + y_diff*y_diff;
	double dist = sqrt(dist_squared);
	double force_coeff = (dist*compulsion_constant)/(lengths[idx]);
	double x_force = x_diff * force_coeff;
	double y_force = y_diff * force_coeff;
	atomicAdd(&(velocities[edge.p1].x), x_force*time_quantum/vertex_mass);
	atomicAdd(&(velocities[edge.p1].y), y_force*time_quantum/vertex_mass);
	atomicAdd(&(velocities[edge.p2].x), -x_force*time_quantum/vertex_mass);
	atomicAdd(&(velocities[edge.p2].y), -y_force*time_quantum/vertex_mass);	
}


__global__ void simulate_repulsion(const Point<double>* points,  Point<double>* velocities, std::size_t point_count, double repulsion_constant, double vertex_mass, double time_quantum){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double f_x = 0;
	double f_y = 0;
	for (int idx2 = 0; idx2 < point_count; idx2++)
	{
		double x_diff = points[idx2].x - points[idx].x;
		double y_diff = points[idx2].y - points[idx].y;
		double dist_squared = x_diff * x_diff + y_diff * y_diff;
		dist_squared = MAX(dist_squared,(double)0.0001);
		double f = repulsion_constant / (dist_squared * sqrt(dist_squared));
		f_y += (f * -y_diff);
		f_x += (f * -x_diff);
	}
	velocities[idx].x+= f_x * time_quantum / vertex_mass;
	velocities[idx].y+= f_y * time_quantum / vertex_mass;
}

__global__ void simulate_movement(Point<double>* points,  Point<double>* velocities, double time_quantum, double slowdown){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	velocities[idx].x *= (slowdown);
	velocities[idx].y *= (slowdown);
	points[idx].x+=velocities[idx].x*time_quantum;
	points[idx].y+=velocities[idx].y*time_quantum;
}

/*
 * This is how a kernel call should be wrapped in a regular function call,
 * so it can be easilly used in cpp-only code.
 */
void step(Point<double>* points, const Edge<std::uint32_t>* edges, const std::uint32_t* lengths, Point<double>* velocities, const  ModelParameters<double>& parameters, std::size_t point_count, std::size_t edge_count)
{
	simulate_repulsion<<<point_count/POINT_BLOCK, POINT_BLOCK>>>(points, velocities, point_count, parameters.vertexRepulsion, parameters.vertexMass, parameters.timeQuantum);
	simulate_compulsion<<<edge_count/EDGE_BLOCK, EDGE_BLOCK>>>(points, edges, lengths, velocities, parameters.edgeCompulsion, parameters.vertexMass, parameters.timeQuantum);
	simulate_movement<<<point_count/POINT_BLOCK, POINT_BLOCK>>>(points, velocities, parameters.timeQuantum, parameters.slowdown);
}
